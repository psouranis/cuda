#include "hip/hip_runtime.h"
#include <stdio.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAException.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define IN_TILE_DIM 5
#define OUT_TILE_DIM ((IN_TILE_DIM) - 2 * (FILTER_RADIUS))
#define FILTER_RADIUS 1

__constant__ float F_c[2*FILTER_RADIUS+1][2*FILTER_RADIUS+1];
__global__ void convolution_cached_tiled_2D_const_mem_kernel(float* N, float*P, int width, int height) {
	int col = blockIdx.x * TILE_DIM + threadIdx.x;
	int row = blockIdx.y * TILE_DIM + threadIdx.y;
	//Loading input tile
	__shared__ float N_s[TILE_DIM][TILE_DIM];
	if (row < height && col < width) {
		N_s[threadIdx.y][threadIdx.x] = N[row * width + col];
	} else {
		N_s[threadIdx.y][threadIdx.x] = 0.0f;
	}
	__syncthreads();
	// Caclulating output elements
	// Turning off the threads at the edges of the block
	if (col < width && row < height) {
		float Pvalue = 0.0f;
		for (int fRow = 0; fRow < 2 * FILTER_RADIUS + 1; fRow++) {
			for (int fCol = 0; fCol < 2 * FILTER_RADIUS + 1; fCol++) {
				if (threadIdx.x - FILTER_RADIUS + fCol >= 0 &&
					threadIdx.x - FILTER_RADIUS + fCol < TILE_DIM &&
					threadIdx.y - FILTER_RADIUS + fRow >=0 &&
					threadIdx.y - FILTER_RADIUS + fRow < TILE_DIM) {
						Pvalue += F_c[fRow][fCol] * N_s[threaIdx.y + fRow][threadIdx.x + fCol];
					}
				else {
					if (row - FILTER_RADIUS + fRow >= 0 &&
						row - FILTER_RADIUS + fRow < height &&
						col - FILTER_RADIUS + fCol >= 0 &&
						col - FILTER_RADIUS + fCol < width) {
							Pvalue += F_c[fRow][fCol] * N[(row-FILTER_RADIUS+fRow)*width + col - FILTER_RADIUS + fCol];
					}
				}
			}
			P[row*width + col] = Pvalue;
		}
	}
}

torch::Tensor conv2d(torch::Tensor matrix) {
    const auto height = matrix.size(0);
    const auto width = matrix.size(1);

    auto result = torch::empty_like(matrix);

    dim3 threads_per_block(IN_TILE_DIM, IN_TILE_DIM); // launches thread blocks whose dimension matches that of the input tiles
    dim3 number_of_blocks((width + threads_per_block.x - 1) / threads_per_block.x,
                          (height + threads_per_block.y - 1) / threads_per_block.y);

    convolution_tiled_2D_const_mem_kernel<<<number_of_blocks, threads_per_block>>>(
        matrix.data_ptr<float>(), result.data_ptr<float>(), width, height);

    return result;
    }